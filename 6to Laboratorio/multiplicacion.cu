
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
using namespace std;

#define TILE_WIDTH 32

void fillMatrix(int* a, int n)
{
   int i;
   for (i = 0; i < n*n; ++i)
        a[i] = 10;//rand()%5;
}

__global__ 
void matrixAdition(int *c, int *a, int *b,int n) 
{
    int ij = threadIdx.x + blockDim.x * blockIdx.x;
		if(ij<(n*n))
			c[ij] = a[ij] + b[ij];
}

__global__ 
void matrixAditionRow(int *c, int *a, int *b,int n) 
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;
   //	if(blockDim.x != 0)
   	//printf("%d  salida\n", ij);
	for(int i =0 ;i<n;i++)
	{
		if(ij<n)
			c[ij*n+i] = a[ij*n+i] + b[ij*n+i];
	}
}

__global__ 
void matrixAditionCol(int *c, int *a, int *b,int n) 
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;
	for(int i =0 ;i<n;i++)
	{
		if(ij<n)
			c[ij+n*i] = a[ij+n*i] + b[ij+n*i];
	}
}
__global__ 
void matrixMulti(int *c, int *a, int *b,int n) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y ; 
    int col = blockIdx.x * blockDim.x + threadIdx.x ;
    if ((row <n) && (col<n))
    {
		int suma=0;
        for(int i=0;i<n;++i)
        {
        	suma+=a[row*n+i]*b[i*n+col];
        }
        c[row*n+col] = suma;	
    } 
}
__global__ void MatrixMulKernel(int * d_P, int * d_M, int* d_N,int Width) 
{
	__shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ int Nds[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	// Identify the row and column of the d_P element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	int Pvalue = 0;
	// Loop over the d_M and d_N tiles required to compute d_P element
	for (int ph = 0; ph < Width/TILE_WIDTH; ++ph) 
	{
		// Collaborative loading of d_M and d_N tiles into shared memory
		Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH + tx];
		Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*Width + Col];
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k)
		{
	 		Pvalue += Mds[ty][k] * Nds[k][tx];
		}
	 	__syncthreads();
	}
	d_P[Row*Width + Col] = Pvalue;
}

void printMatrix(string s, int *a , int tam){
	cout<<s;
	for(int i=0;i<tam;i++)
	{
		for(int j=0;j<tam;j++)
		{
			cout<<a[i*tam+j]<<" ";
		}
		cout<<endl;
	}
}

int main(int argc, char *argv[])
{
	srand (time(NULL));
	int  N= strtol(argv[1], NULL, 10);
	//cout<<N<<endl; return 1;
	int THREADS_PER_BLOCK =32;	
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; //device copies of a,b,c
	//int size = N*N*sizeof(int);
	int size=N*N*sizeof(int);
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); 
	fillMatrix(a, N);
	b = (int *)malloc(size); 
	fillMatrix(b, N);
	c = (int *)malloc(size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	/*int blocks= (N*N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	int blocks2= (N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	cout<<"blocks : \n"<<blocks2<<"\n threds: \n "<<THREADS_PER_BLOCK<<endl; */

	int blocks= (N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	dim3 dimGrid(blocks, blocks, 1);
	dim3 dimBlock(THREADS_PER_BLOCK,THREADS_PER_BLOCK, 1);
	cout<<"blocks : \n"<<blocks<<"\n threds: \n "<<THREADS_PER_BLOCK<<endl; 
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventRecord(start,0);
		//matrixAdition<<<blocks,THREADS_PER_BLOCK>>>( d_c, d_a, d_b,N);
		//matrixAditionRow<<<blocks2,THREADS_PER_BLOCK>>>(d_c, d_a, d_b,N);
		//matrixMulti<<<blocks2,THREADS_PER_BLOCK>>>(d_c, d_a, d_b,N);
		//matrixAditionCol<<<blocks2,THREADS_PER_BLOCK>>>( d_c, d_a, d_b,N);
		matrixMulti<<<dimGrid,dimBlock>>>(d_c, d_a, d_b, N);
		//MatrixMulKernel<<<dimGrid,dimBlock>>>(d_c, d_a, d_b, N);
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);
	printf("Elapsed time : %f ms\n" ,elapsedTime);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	//printMatrix("Printing Matrix A \n",a,N);
	//printMatrix("Printing Matrix B \n",b,N);
	//printMatrix("Printing Matrix C \n",c,N);
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
