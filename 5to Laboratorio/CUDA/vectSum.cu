// Compute vector sum h_C = h_A+h_B
void vecAdd(float* h_A, float* h_B, float* h_C, int n)
{
	for (i = 0; i < n; i++) 
		h_C[i] = h_A[i] + h_B[i];
}
int main()
{
	// Memory allocation for h_A, h_B, and h_C
	// I/O to read h_A and h_B, N elements each
	…
	vecAdd(h_A, h_B, h_C, N);
}