
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <cmath>
#include <string>

using namespace std;
#define CHANNELS 3
#define BLUR_SIZE 1

void print_vect(int *V, int n){
    int i;
    for (i = 0; i < n; i++)
		printf("%d ", V[i]);
}

int* ReadSizeImg(char * name)
{
    int * dim= new int[2];
    int fil, col;
    string line,s1;
    ifstream file(name);
    getline(file,line);
    getline(file,line);
    file>>fil>>col;
    dim[0]=fil; dim[1]=col;   
    return dim;
}
void ReadPPM(int *Pin, char *name)
{
	int e1;
    string line,s1;
    ifstream file(name);
    getline(file,line);
    getline(file,line);
    getline(file,line);
    getline(file,line);
    int m=0;
    while(!file.eof())
    {
        file>>e1;
       // cout<<e1<<endl;
        //if(!e1) break;
        Pin[m]=e1;
        m++;
    }
}

void WritePGM(int * Pout, int fil , int cols, char *name)
{
    ofstream file(name);
    file<<"P2"<<endl;
    file<<"# Function ConvertRGBtoGray @eddyrene"<<endl;
    file<<fil<<" "<<cols<<endl;
    file<<255<<endl;
    int n = fil*cols;
    int i=0;
    while(i<n)
    {
        file<<Pout[i]<<endl;
        i++;
    }
}

void WritePPM(int * Pout, int fil , int cols, char *name)
{
    ofstream file(name);
    file<<"P3"<<endl;
    file<<"# Function ConvertRGBtoGray @eddyrene"<<endl;
    file<<fil<<" "<<cols<<endl;
    file<<255<<endl;
    int n = fil*cols;
    int i=0;
    while(i<3*n)
    {
        file<<Pout[i]<<endl;
        i++;
    }
}


// we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__
void colorToGreyscaleConversion(int * Pout, int * Pin, int width, int height) 
{
	int Col = threadIdx.x + blockIdx.x * blockDim.x;
	int Row = threadIdx.y + blockIdx.y * blockDim.y;
	if (Col < width && Row < height) {
		// get 1D coordinate for the grayscale image
		int greyOffset = Row*width + Col;
		// one can think of the RGB image having
		// CHANNEL times columns than the grayscale image
		int rgbOffset = greyOffset*CHANNELS;
		int r = Pin[rgbOffset ]; // red value for pixel
		int g = Pin[rgbOffset + 2]; // green value for pixel
		int b = Pin[rgbOffset + 3]; // blue value for pixel
		// perform the rescaling and store it
		// We multiply by floating point constants
		Pout[greyOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	}
}

__global__
void blurKernel(int * in, int * out, int w, int h) 
{
	int Col = blockIdx.x * blockDim.x + threadIdx.x;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	if (Col < w && Row < h) {
		int pixVal = 0;
		int pixels = 0;
		// Get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
		for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
			for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {
				int curRow = Row + blurRow;
				int curCol = Col + blurCol;
				// Verify we have a valid image pixel
				if(curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
					pixVal += in[curRow * w + curCol];
					pixels++; // Keep track of number of pixels in the accumulated total
				}
			}
		}
		// Write our new pixel value out
		printf("Esperado %d  %d  %d \n",pixVal,pixels , pixVal / (float)pixels );
		out[Row * w + Col] = (int)(pixVal / (float)pixels);
	}
}

int main()
{
	int * Pin,* Pout;
    int * d_Pin,* d_Pout;
	int * order = ReadSizeImg("lena.ppm");
	int N=order[0]; int M=order[1];
	int THREADS_PER_BLOCK = 16
;    int size = 3*N*M*sizeof(int);

    cout<<"tamano Imagen "<<N<<" "<<M<<"  size "<<size<<endl;

    hipMalloc((void **)&d_Pin, size);
	hipMalloc((void **)&d_Pout, size);
    cout<<"pasa"<<endl;
    Pin = (int *)malloc(size); 		ReadPPM(Pin,"lena.ppm");
    //print_vect(Pin,3*order[0]*order[1]);
    Pout = (int *)malloc(size); 

	hipMemcpy(d_Pin, Pin, size, hipMemcpyHostToDevice);
	hipMemcpy(d_Pout, Pout, size, hipMemcpyHostToDevice);

	int blocks= (N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	dim3 dimGrid(blocks, blocks, 1);
	dim3 dimBlock(THREADS_PER_BLOCK,THREADS_PER_BLOCK, 1);
	cout<<"blocks : \n"<<blocks<<"\n threds: \n "<<THREADS_PER_BLOCK<<endl; 
		colorToGreyscaleConversion<<<dimGrid,dimBlock>>>( d_Pout, d_Pin, N, M);
		//blurKernel<<<dimGrid,dimBlock>>>( d_Pout, d_Pin, N, M);
	hipMemcpy(Pout, d_Pout, size, hipMemcpyDeviceToHost);
	WritePGM(Pout, N,M,"lenaInGray.ppm");  
	//WritePPM(Pout, N,M,"n.ppm");  
	free(Pin); free(Pout);
	hipFree(d_Pin); hipFree(d_Pout);
	return 0;
}

    
