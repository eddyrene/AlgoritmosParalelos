
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <cmath>
#include <string>
using namespace std;

//#define THREADS_PER_BLOCK 32

void fillMatrix(int* a, int n)
{
   int i;
   for (i = 0; i < n*n; ++i)
        a[i] = 10;//rand()%5;
}

__global__ 
void matrixAdition(int *c, int *a, int *b,int n) 
{
    int ij = threadIdx.x + blockDim.x * blockIdx.x;
		if(ij<(n*n))
			c[ij] = a[ij] + b[ij];
}

__global__ 
void matrixAditionRow(int *c, int *a, int *b,int n) 
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;
   //	if(blockDim.x != 0)
   	//printf("%d  salida\n", ij);
	for(int i =0 ;i<n;i++)
	{
		if(ij<n)
			c[ij*n+i] = a[ij*n+i] + b[ij*n+i];
	}
}

__global__ 
void matrixAditionCol(int *c, int *a, int *b,int n) 
{
   	int ij = threadIdx.x + blockDim.x * blockIdx.x;
	for(int i =0 ;i<n;i++)
	{
		if(ij<n)
			c[ij+n*i] = a[ij+n*i] + b[ij+n*i];
	}
}

void printMatrix(string s, int *a , int tam){
	cout<<s;
	for(int i=0;i<tam;i++)
	{
		for(int j=0;j<tam;j++)
		{
			cout<<a[i*tam+j]<<" ";
		}
		cout<<endl;
	}
}

int main(int argc, char *argv[])
{
	srand (time(NULL));
	int  N= strtol(argv[1], NULL, 10);
	//cout<<N<<endl; return 1;
	int THREADS_PER_BLOCK =1024;	
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; //device copies of a,b,c
	//int size = N*N*sizeof(int);
	//allocateMemory(a,b,c,d_a,d_b,d_c,N);
	int size=N*N*sizeof(int);
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); 
	fillMatrix(a, N);
	b = (int *)malloc(size); 
	fillMatrix(b, N);
	c = (int *)malloc(size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	int blocks= (N*N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	int blocks2= (N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	cout<<"blocks : \n"<<blocks2<<"\n threds: \n "<<THREADS_PER_BLOCK<<endl; 
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventRecord(start,0);
		//matrixAdition<<<blocks,THREADS_PER_BLOCK>>>( d_c, d_a, d_b,N);
		matrixAditionRow<<<blocks2,THREADS_PER_BLOCK>>>( d_c, d_a, d_b,N);
		//matrixAditionCol<<<blocks2,THREADS_PER_BLOCK>>>( d_c, d_a, d_b,N);
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);
	printf("Elapsed time : %f ms\n" ,elapsedTime);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	//printMatrix("Printing Matrix A \n",a,N);
	//printMatrix("Printing Matrix B \n",b,N);
	//printMatrix("Printing Matrix C \n",c,N);
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
